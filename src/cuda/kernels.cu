
#include <hip/hip_runtime.h>
extern "C" {
    __global__ void fill_u8(unsigned char *y, unsigned char elem, unsigned int len) {
        int tid = blockIdx.x * blockDim.x + threadIdx.x; 
        if (tid < len) {
            y[tid] = elem;
        }
    }
    __global__ void fill_u32(unsigned int *y, unsigned int elem, unsigned int len) {
        int tid = blockIdx.x * blockDim.x + threadIdx.x; 
        if (tid < len) {
            y[tid] = elem;
        }
    }
    __global__ void u8_to_f32(const unsigned char* x, float* y, unsigned int len) {
        const float scale = 1.0f / 255.0f;
        int tid = blockIdx.x * blockDim.x + threadIdx.x; 
        if (tid < len) {
            y[tid] = scale * x[tid];
        }
    }
    __global__ void u8_to_one_hot_f32(const unsigned char* x, unsigned int nclasses, float* y, unsigned int len) {
        int tid = blockIdx.x * blockDim.x + threadIdx.x; 
        if (tid < len) {
            y[tid*nclasses+x[tid]] = 1.0f;
        }
    } 
    __global__ void add(const float* x1, const float* x2, float* y, unsigned int len) {
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        if (tid < len) {
            y[tid] = x1[tid] + x2[tid];
        }
    } 
    __global__ void cross_entropy_forward(unsigned int batch_size, unsigned int nclasses, const float* x, const float* t, float* y) {
        int tid = blockIdx.x * blockDim.x + threadIdx.x; 
        if (tid < batch_size) {
            // compute max value of slice
            float m = x[tid*nclasses];
            for(int i = 1; i < nclasses; ++i) {
                m = fmaxf(x[tid*nclasses+i], m);
            } 
            // subtract max
            for(int i = 0; i < nclasses; ++i) {
              y[tid*nclasses+i] = x[tid*nclasses+i]-m;
            }
            // sum
            float s = 0.0f;
            for(int i = 0; i < nclasses; ++i) {
              s += expf(y[tid*nclasses+i]);
            }
            // compute ln(s)
            float ln_s = logf(s);
            // y = (ln_s - y) * t
            for(int i = 0; i < nclasses; ++i) {
                y[tid*nclasses+i] = (ln_s - y[tid*nclasses+i]) * t[tid*nclasses+i];
            }
        }
    }
    __global__ void cross_entropy_backward(const float* x, float* dx, const float* t, float* dy, unsigned int len) {
        int tid = blockIdx.x * blockDim.x + threadIdx.x; 
        if (tid < len) {
            dx[tid] = dy[0] * (x[tid] - t[tid]);
        }
    }
    __global__ void reduce_sum_partial(const float* input, float* output, unsigned int len) {
        // from http://www.techdarting.com/2014/06/parallel-reduction-in-cuda.html
        // Load a segment of the input vector into shared memory
        __shared__ float partialSum[2*256];
        int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
        unsigned int t = threadIdx.x;
        unsigned int start = 2*blockIdx.x*blockDim.x;

        if ((start + t) < len)
        {
            partialSum[t] = input[start + t];      
        }
        else
        {       
            partialSum[t] = 0.0;
        }
        if ((start + blockDim.x + t) < len)
        {   
            partialSum[blockDim.x + t] = input[start + blockDim.x + t];
        }
        else
        {
            partialSum[blockDim.x + t] = 0.0;
        }

        // Traverse reduction tree
        for (unsigned int stride = blockDim.x; stride > 0; stride /= 2)
        {
          __syncthreads();
            if (t < stride)
                partialSum[t] += partialSum[t + stride];
        }
        __syncthreads();

        // Write the computed sum of the block to the output vector at correct index
        if (t == 0 && (globalThreadId*2) < len)
        {
            output[blockIdx.x] = partialSum[t];
        }
        }
        __global__ void reduce_sum_final(const float* x, float* y, unsigned int len) {
        *y = 0;
        for(int i = 0; i < len; ++i) {
          *y += x[i];  
        }
    }
    __global__ void reverse_conv_filter(const float* x, float beta, float* y, unsigned int filter_len, unsigned int len) {
        int tid = blockIdx.x*blockDim.x + threadIdx.x;
        if (tid < len) {
            if (beta == 0.0f) {
                for(int i = 0; i < filter_len; ++i) {
                    y[tid*filter_len + i] = x[tid*filter_len + ((filter_len-1) - i)];
                }
            }
          else {
              for(int i = 0; i < filter_len; ++i) {
                y[tid*filter_len + i] = x[tid*filter_len + ((filter_len-1) - i)] + beta * y[tid*filter_len + i];
              }
          }
        }
    }
    __global__ void sgd_with_momentum(float* w, const float* dw, float learning_rate, float momentum, float* v, unsigned int len) {
        int tid = blockIdx.x*blockDim.x + threadIdx.x;
        if (tid < len) {
            v[tid] = momentum * v[tid] + dw[tid];
            w[tid] -= learning_rate * v[tid];
        }
    }
}
